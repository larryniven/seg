#include "hip/hip_runtime.h"
#include "scrf/util.h"
#include "scrf/scrf.h"
#include "scrf/lm.h"
#include "scrf/lattice.h"
#include "speech/speech.h"
#include "scrf/loss.h"
#include "scrf/scrf_cost.h"
#include "scrf/scrf_feat.h"
#include "scrf/scrf_weight.h"
#include "scrf/scrf_util.h"
#include "scrf/make_feat.h"
#include <fstream>

#include "nn/nn-gpu.h"
#include "autodiff/autodiff-gpu.h"
#include "scrf/e2e-util.h"

struct learning_env {

    std::ifstream frame_batch;
    std::ifstream ground_truth_batch;
    std::shared_ptr<lm::fst> lm;
    int min_seg;
    int max_seg;

    scrf::param_t param;
    scrf::param_t opt_data;

    double step_size;
    double nn_step_size;

    int save_every;

    nn::gpu::param_t nn_param;
    nn::gpu::opt_t nn_opt_data;
    nn::nn_t nn;

    std::string output_param;
    std::string output_opt_data;

    std::string output_nn_param;
    std::string output_nn_opt_data;

    std::vector<std::string> features;

    std::unordered_map<std::string, int> phone_id;

    std::unordered_map<std::string, std::string> args;

    learning_env(std::unordered_map<std::string, std::string> args);

    void run();

};

nn::gpu::param_t nn_backprop(std::unordered_map<int, la::vector<double>> const& grad,
    std::vector<std::vector<double>> const& frames, nn::nn_t& nn, nn::gpu::param_t const& param);

struct hinge_loss {

    fst::path<scrf::scrf_t> const& gold_path;
    fst::path<scrf::scrf_t> cost_aug_path;
    std::unordered_map<std::string, int> const& phone_id;

    hinge_loss(fst::path<scrf::scrf_t> const& gold_path,
        scrf::scrf_t const& graph,
        std::unordered_map<std::string, int> const& phone_id);

    double loss() const;

    std::unordered_map<int, la::vector<double>> feat_grad() const;

    scrf::param_t param_grad() const;

};

int main(int argc, char *argv[])
{
    ebt::ArgumentSpec spec {
        "learn-e2e",
        "Learn segmental CRF and neural network end to end",
        {
            {"frame-batch", "", true},
            {"ground-truth-batch", "", true},
            {"lm", "", true},
            {"min-seg", "", false},
            {"max-seg", "", false},
            {"min-cost-path", "Use min cost path for training", false},
            {"param", "", true},
            {"opt-data", "", true},
            {"step-size", "", true},
            {"nn-step-size", "", true},
            {"save-every", "", false},
            {"nn-param", "", true},
            {"nn-opt-data", "", true},
            {"output-param", "", false},
            {"output-opt-data", "", false},
            {"output-nn-param", "", false},
            {"output-nn-opt-data", "", false},
            {"phone-id", "", true},
            {"features", "", true}
        }
    };

    if (argc == 1) {
        ebt::usage(spec);
        exit(1);
    }

    auto args = ebt::parse_args(argc, argv, spec);

    std::cout << args << std::endl;

    learning_env env { args };

    env.run();

    return 0;
}

learning_env::learning_env(std::unordered_map<std::string, std::string> args)
    : args(args)
{
    if (ebt::in(std::string("frame-batch"), args)) {
        frame_batch.open(args.at("frame-batch"));
    }

    ground_truth_batch.open(args.at("ground-truth-batch"));

    lm = std::make_shared<lm::fst>(lm::load_arpa_lm(args.at("lm")));

    min_seg = 1;
    if (ebt::in(std::string("min-seg"), args)) {
        min_seg = std::stoi(args.at("min-seg"));
    }

    max_seg = 20;
    if (ebt::in(std::string("max-seg"), args)) {
        max_seg = std::stoi(args.at("max-seg"));
    }

    param = scrf::load_param(args.at("param"));
    step_size = std::stod(args.at("step-size"));
    nn_step_size = std::stod(args.at("nn-step-size"));

    if (ebt::in(std::string("save-every"), args)) {
        save_every = std::stoi(args.at("save-every"));
    } else {
        save_every = std::numeric_limits<int>::max();
    }

    nn_param = nn::load_param(args.at("nn-param"));
    nn_opt_data = nn::load_opt(args.at("nn-opt-data"));

    output_param = "param-last";
    if (ebt::in(std::string("output-param"), args)) {
        output_param = args.at("output-param");
    }

    output_opt_data = "opt-data-last";
    if (ebt::in(std::string("output-opt-data"), args)) {
        output_opt_data = args.at("output-opt-data");
    }

    output_nn_param = "nn-param-last";
    if (ebt::in(std::string("output-nn-param"), args)) {
        output_nn_param = args.at("output-nn-param");
    }

    output_nn_opt_data = "nn-opt-data-last";
    if (ebt::in(std::string("output-nn-opt-data"), args)) {
        output_nn_opt_data = args.at("output-nn-opt-data");
    }

    phone_id = scrf::load_phone_id(args.at("phone-id"));

    features = ebt::split(args.at("features"), ",");
}

void learning_env::run()
{
    std::shared_ptr<lm::fst> lm_output = scrf::erase_input(lm);

    int i = 1;

    while (1) {
        nn = nn::gpu::make_nn(nn_param);

        std::vector<std::vector<real>> frames;

        frames = speech::load_frame_batch(frame_batch);

        lattice::fst ground_truth_lat = lattice::load_lattice(ground_truth_batch);

        if (!frame_batch || !ground_truth_batch) {
            break;
        }

        std::vector<std::vector<real>> inputs = scrf::nn_feedforward(frames, nn);

        std::cout << ground_truth_lat.data->name << std::endl;

        std::cout << "ground truth: ";
        for (auto& e: ground_truth_lat.edges()) {
            std::cout << ground_truth_lat.output(e) << " ";
        }
        std::cout << std::endl;

        scrf::scrf_t ground_truth = scrf::make_gold_scrf(ground_truth_lat, lm);
        fst::path<scrf::scrf_t> ground_truth_path = scrf::make_ground_truth_path(ground_truth);

        scrf::scrf_t min_cost = scrf::make_graph_scrf(frames.size(), lm_output, min_seg, max_seg);

        scrf::scrf_t gold;
        fst::path<scrf::scrf_t> gold_path;

        if (ebt::in(std::string("min-cost-path"), args)) {
            gold = min_cost;
            gold_path = scrf::make_min_cost_path(min_cost, ground_truth_path);
        } else {
            gold = ground_truth;
            gold_path = ground_truth_path;
        }
        gold_path.data->base_fst = &gold;

        scrf::composite_feature gold_feat_func = scrf::make_feat(features, inputs, phone_id);

        gold.weight_func = std::make_shared<scrf::score::linear_score>(
            scrf::score::linear_score(param, std::make_shared<scrf::composite_feature>(gold_feat_func)));
        gold.feature_func = std::make_shared<scrf::composite_feature>(gold_feat_func);

        scrf::composite_feature graph_feat_func = scrf::make_feat(features, inputs, phone_id);

        scrf::scrf_t graph = scrf::make_graph_scrf(frames.size(), lm_output, min_seg, max_seg);

        scrf::seg_cost cost = scrf::make_overlap_cost(gold_path);
        scrf::score::linear_score score { param, std::make_shared<scrf::composite_feature>(graph_feat_func) };

        graph.weight_func = std::make_shared<scrf::score::linear_score>(score)
            + std::make_shared<scrf::seg_cost>(cost);
        graph.feature_func = std::make_shared<scrf::composite_feature>(graph_feat_func);

        hinge_loss loss_func { gold_path, graph, phone_id };
        real ell = loss_func.loss();

        std::cout << "gold segs: " << gold_path.edges().size()
            << " frames: " << frames.size() << std::endl;
        std::cout << "loss: " << ell << std::endl;

        if (ell < -1e6) {
            std::cerr << "weird loss value. exit." << std::endl;
            exit(1);
        }

        if (ell < 0) {
            std::cout << "loss is less than zero.  skipping." << std::endl;
        }

#if 0
        {
            nn::param_t nn_param2 = nn_param;
            nn_param2.label_bias(phone_id.at("<s>")) += 1e-8;
            nn::nn_t nn2 = nn::make_nn(nn_param2);

            std::vector<std::vector<real>> inputs2 = scrf::nn_feedforward(frames, nn2);

            scrf::scrf_t ground_truth = scrf::make_gold_scrf(ground_truth_lat, lm);
            fst::path<scrf::scrf_t> ground_truth_path = scrf::make_ground_truth_path(ground_truth);

            scrf::scrf_t min_cost = scrf::make_graph_scrf(frames.size(), lm_output, min_seg, max_seg);

            scrf::scrf_t gold;
            fst::path<scrf::scrf_t> gold_path;

            if (ebt::in(std::string("min-cost-path"), args)) {
                gold = min_cost;
                gold_path = scrf::make_min_cost_path(min_cost, ground_truth_path);
            } else {
                gold = ground_truth;
                gold_path = ground_truth_path;
            }
            gold_path.data->base_fst = &gold;

            scrf::composite_feature gold_feat_func { inputs2, phone_id };

            gold.weight_func = std::make_shared<scrf::score::linear_score>(
                scrf::score::linear_score(param, std::make_shared<scrf::composite_feature>(gold_feat_func)));
            gold.feature_func = std::make_shared<scrf::composite_feature>(gold_feat_func);

            scrf::composite_feature graph_feat_func { inputs2, phone_id };

            scrf::scrf_t graph = scrf::make_graph_scrf(frames.size(), lm_output, min_seg, max_seg);

            scrf::seg_cost cost = scrf::make_overlap_cost(gold_path);
            scrf::score::linear_score score { param, std::make_shared<scrf::composite_feature>(graph_feat_func) };

            graph.weight_func = std::make_shared<scrf::score::linear_score>(score)
                + std::make_shared<scrf::seg_cost>(cost);
            graph.feature_func = std::make_shared<scrf::composite_feature>(graph_feat_func);

            hinge_loss loss_func2 { gold_path, graph, phone_id };

            {
                auto edges1 = loss_func.gold_path.edges();
                auto edges2 = loss_func2.gold_path.edges();

                for (int i = 0; i < edges1.size(); ++i) {
                    int tail = std::get<0>(loss_func.gold_path.tail(edges1[i]));
                    int head = std::get<0>(loss_func.gold_path.head(edges1[i]));

                    auto& lat = *loss_func.gold_path.data->base_fst->fst->fst1;

                    std::cout << edges1[i] << " " << edges2[i] << " "
                        << lat.data->vertices.at(tail).time << " "
                        << lat.data->vertices.at(head).time << " "
                        << loss_func.gold_path.output(edges1[i]) << " "
                        << loss_func2.gold_path.weight(edges1[i]) << " "
                        << loss_func.gold_path.weight(edges2[i]) << " "
                        << loss_func2.gold_path.weight(edges1[i]) - loss_func.gold_path.weight(edges2[i])
                        << std::endl;
                }
            }

            {
                auto edges1 = loss_func.cost_aug_path.edges();
                auto edges2 = loss_func2.cost_aug_path.edges();

                for (int i = 0; i < edges1.size(); ++i) {
                    int tail = std::get<0>(loss_func.gold_path.tail(edges1[i]));
                    int head = std::get<0>(loss_func.gold_path.head(edges1[i]));

                    auto& lat = *loss_func.gold_path.data->base_fst->fst->fst1;

                    std::cout << edges1[i] << " " << edges2[i] << " "
                        << lat.data->vertices.at(tail).time << " "
                        << lat.data->vertices.at(head).time << " "
                        << loss_func2.gold_path.weight(edges1[i]) << " "
                        << loss_func.gold_path.weight(edges2[i]) << " "
                        << loss_func2.gold_path.weight(edges1[i]) - loss_func.gold_path.weight(edges2[i])
                        << std::endl;
                }
            }

            double ell2 = loss_func2.loss();

            std::cout << "numeric grad: " << (ell2 - ell) / 1e-8
                << " " << ell2 << " " << ell << " " << ell2 - ell << std::endl;
        }
#endif

        if (ell > 0) {
            std::unordered_map<int, la::vector<double>> feat_grad = loss_func.feat_grad();
            nn::gpu::param_t nn_grad = nn_backprop(feat_grad, frames, nn, nn_param);

            // std::cout << "calc grad: " << nn_grad.label_bias(phone_id.at("<s>")) << std::endl;

            nn::gpu::adagrad_update(nn_param, nn_grad, nn_opt_data, nn_step_size);

            scrf::param_t grad = loss_func.param_grad();
            scrf::adagrad_update(param, grad, opt_data, step_size);

            if (i % save_every == 0) {
                scrf::save_param(param, "param-last");
                scrf::save_param(opt_data, "opt-data-last");

                nn::save_param(nn::gpu::to_host(nn_param), "nn-param-last");
                nn::save_opt(nn::gpu::to_host(nn_opt_data), "nn-opt-data-last");
            }
        }

        std::cout << std::endl;

#if DEBUG_TOP_10
        if (i == 10) {
            break;
        }
#endif

        ++i;
    }

    scrf::save_param(param, output_param);
    scrf::save_param(opt_data, output_opt_data);

    nn::save_param(nn::gpu::to_host(nn_param), output_nn_param);
    nn::save_opt(nn::gpu::to_host(nn_opt_data), output_nn_opt_data);

}

hinge_loss::hinge_loss(
    fst::path<scrf::scrf_t> const& gold_path,
    scrf::scrf_t const& graph,
    std::unordered_map<std::string, int> const& phone_id)
    : gold_path(gold_path), phone_id(phone_id)
{
    auto order = scrf::topo_order(graph);
    cost_aug_path = scrf::shortest_path(graph, order);
}

double hinge_loss::loss() const
{
    double gold_score = 0;
    for (auto& e: gold_path.edges()) {
        gold_score += gold_path.weight(e);
    }

    std::cout << "gold score: " << gold_score << std::endl;

    scrf::seg_cost cost = scrf::make_overlap_cost(gold_path);

    double cost_aug_weight = 0;
    double cost_aug_cost = 0;
    std::cout << "cost aug path: ";
    for (auto& e: cost_aug_path.edges()) {
        cost_aug_weight += cost_aug_path.weight(e);
        cost_aug_cost += cost(*cost_aug_path.data->base_fst->fst, e);
        std::cout << cost_aug_path.output(e) << " ";
    }
    std::cout << std::endl;
    std::cout << "cost aug cost: " << cost_aug_cost << std::endl;
    std::cout << "cost aug score: " << cost_aug_weight - cost_aug_cost << std::endl;

    return cost_aug_weight - gold_score;
}

std::unordered_map<int, la::vector<double>> hinge_loss::feat_grad() const
{
    std::unordered_map<int, la::vector<double>> result;

    lattice::fst& gold_lat = *gold_path.data->base_fst->fst->fst1;
    for (auto& e: gold_path.edges()) {
        int tail = std::get<0>(gold_path.tail(e));
        int tail_time = gold_lat.data->vertices.at(tail).time;

        // result[tail_time].resize(phone_id.size());
        // result[tail_time](phone_id.at(gold_path.output(e))) -= 1;

        int head = std::get<0>(gold_path.head(e));
        int head_time = gold_lat.data->vertices.at(head).time;

        // result[head_time - 1].resize(phone_id.size());
        // result[head_time - 1](phone_id.at(gold_path.output(e))) -= 1;

        // result[int((tail_time + head_time - 1) / 2)].resize(phone_id.size());
        // result[int((tail_time + head_time - 1) / 2)](phone_id.at(gold_path.output(e))) -= 1;

        for (int i = tail_time; i < head_time; ++i) {
            result[i].resize(phone_id.size());
            result[i](phone_id.at(gold_path.output(e))) -= 1;
        }
    }

    lattice::fst& cost_aug_lat = *cost_aug_path.data->base_fst->fst->fst1;
    for (auto& e: cost_aug_path.edges()) {
        int tail = std::get<0>(cost_aug_path.tail(e));
        int tail_time = cost_aug_lat.data->vertices.at(tail).time;

        // result[tail_time].resize(phone_id.size());
        // result[tail_time](phone_id.at(cost_aug_path.output(e))) += 1;

        int head = std::get<0>(cost_aug_path.head(e));
        int head_time = cost_aug_lat.data->vertices.at(head).time;

        // result[head_time - 1].resize(phone_id.size());
        // result[head_time - 1](phone_id.at(cost_aug_path.output(e))) += 1;

        // result[int((tail_time + head_time - 1) / 2)].resize(phone_id.size());
        // result[int((tail_time + head_time - 1) / 2)](phone_id.at(cost_aug_path.output(e))) += 1;

        for (int i = tail_time; i < head_time; ++i) {
            result[i].resize(phone_id.size());
            result[i](phone_id.at(cost_aug_path.output(e))) += 1;
        }
    }

    return result;
}

scrf::param_t hinge_loss::param_grad() const
{
    scrf::param_t result;

    scrf::scrf_t const& gold_scrf = *gold_path.data->base_fst;
    for (auto& e: gold_path.edges()) {
        scrf::feat_t f;
        (*gold_scrf.feature_func)(f, *gold_scrf.fst, e);
        scrf::isub(result, to_param(std::move(f)));
    }

    scrf::scrf_t const& graph_scrf = *cost_aug_path.data->base_fst;
    for (auto& e: cost_aug_path.edges()) {
        scrf::feat_t f;
        (*graph_scrf.feature_func)(f, *graph_scrf.fst, e);
        scrf::iadd(result, to_param(std::move(f)));
    }

    return result;
}

nn::gpu::param_t nn_backprop(std::unordered_map<int, la::vector<double>> const& grad,
    std::vector<std::vector<double>> const& frames, nn::nn_t& nn, nn::gpu::param_t const& param)
{
    int dim = frames.front().size();
    nn::gpu::param_t result;
    nn::gpu::resize_as(result, param);

    la::vector<double> input_block;
    input_block.resize(frames.size() * 11 * dim);

    for (auto& p: grad) {
        int i = p.first;

        std::vector<double> input;

        for (int j = i - 5; j <= i + 5; ++j) {
            if (j < 0 || j >= frames.size()) {
                input.resize(input.size() + dim);
            } else {
                input.insert(input.end(), frames[j].begin(), frames[j].end());
            }
        }

        std::copy(input.begin(), input.end(), input_block.data() + 11 * dim * i);
    }

    la::gpu::vector<double> input_gpu_block { input_block };

    for (auto& p: grad) {
        int i = p.first;

        nn.hidden[0]->output = std::make_shared<la::gpu::vector_view<double>>(
            la::gpu::vector_view<double>(input_gpu_block.data() + 11 * dim * i, 11 * dim));

        autodiff::eval(nn.output, autodiff::gpu::eval_funcs);

        nn.output->grad = std::make_shared<la::gpu::vector<double>>(
            la::gpu::vector<double>(p.second));

        autodiff::grad(nn.output, autodiff::gpu::grad_funcs);

        nn::gpu::iadd(result, nn::gpu::copy_grad(nn));

        nn::gpu::zero_grad(nn);
    }

    return result;
}

